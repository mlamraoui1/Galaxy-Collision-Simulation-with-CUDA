#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>
#include <math.h>

__device__ float4 Interaction(float4 p_i, float4 p_j, float4 a_i)
{
	float4 D;

	D.x = p_j.x - p_i.x;
	D.y = p_j.y - p_i.y;
	D.z = p_j.z - p_i.z;

	float dist = sqrtf(D.x * D.x + D.y * D.y + D.z * D.z);
	

	float dist3 = dist * dist * dist;

	if (dist3 < 1.0f) {
		dist3= 1.0f;
	}
	
	float fact = 10.0f * p_j.w / dist3;

	a_i.x += D.x * fact ;
	a_i.y += D.y * fact ;
	a_i.z += D.z * fact ;

	return a_i;
} 

__device__ float4 tile_calculation(float4 myPosition, float4 acc)
{
	extern __shared__ float4 shPosition[];

	for (unsigned int i = 0; i < BSIZE; i++)
		acc = Interaction(myPosition, shPosition[i], acc);

	return acc;
}


__global__ void galaxyKernel(float4* pos, float4 * pdata, unsigned int width, unsigned int height)
{

	// p bodysdescriptions from the shared memory
	extern __shared__ float4 shPosition[];









__global__ void kernel_saxpy( int n, float a, float * x, float * y, float * z ) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) { 
		z[i] = a * x[i] + y [i];
	}
}

void saxpy( int nblocks, int nthreads, int n, float a, float * x, float * y, float * z ) {
	kernel_saxpy<<<nblocks, nthreads>>>( n, a, x, y, z );
}
